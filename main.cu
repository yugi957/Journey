#include "hip/hip_runtime.h"
//#include "kernels.cuh"
//#include "general.h"
//#include "CUDAFunctions.cuh";
//#include "BMP.h"
//#include "Image.h"
//#include "Image.cuh"
#include "MLP.h"
//#include "matConv.h"
#include <fstream>
#include "MNIST.h"
#include "MLP.cuh"
#include "general.cuh"
#include "general.h"
using namespace std;

int main() {

	
    vector<vector<double>> train_imgs, train_lbls, test_imgs, test_lbls;

    getMNIST(&train_imgs, &train_lbls, &test_imgs, &test_lbls);
	
    int s = train_imgs[0].size();
	int batchSize = 20;
	cout << "\n\n----------IMAGE CLASSIFIER----------\n\n";

	printf("Size: %d\n", train_imgs[0].size());
	//MultiLayerPerceptron* mlp = new MultiLayerPerceptron({s}, CROSS_ENTROPY, 1, .001);
	//mlp->addLayer(512, SIGMOID);
	//mlp->addLayer(512, SIGMOID);
	//mlp->addLayer(10, SOFTMAX);
	MultiLayerPerceptron* mlp = new MultiLayerPerceptron({ s }, CROSS_ENTROPY, 1, .01, batchSize);
	mlp->addLayer(512, SIGMOID);
	mlp->addLayer(512, SIGMOID);
	mlp->addLayer(10, SOFTMAX);
	MultiLayerParatron* mlpara = new MultiLayerParatron({ s }, CROSS_ENTROPY, 1, .01, batchSize);
	mlpara->addLayer(512, SIGMOID);
	mlpara->addLayer(512, SIGMOID);
	mlpara->addLayer(10, SOFTMAX);
	MultiLayerParatron* mlpar = new MultiLayerParatron({ s }, CROSS_ENTROPY, 1, .01, batchSize);
	mlpar->addLayer(512, SIGMOID);
	mlpar->addLayer(512, SIGMOID);
	mlpar->addLayer(10, SOFTMAX);
	for (int i = 0;i < mlp->h_weights.size();i++) {
		for (int j = 0;j < mlp->h_weights[i].size();j++) {
			for (int k = 0;k < mlp->h_weights[i][j].size();k++) {
				mlpara->h_weights[i][j][k] = mlp->h_weights[i][j][k];
				mlpar->h_weights[i][j][k] = mlp->h_weights[i][j][k];
			}
		}
	}

	mlpara->finalize();
	mlpar->finalize();
	mlp->finalize();
	cout << "Training Neural Network as Image Classifier...\n";
	double loss = 0.0;
	int numBatches = train_imgs.size() / batchSize;


    printf("Training on %d images and %d labels...\n", train_imgs.size(), train_lbls.size());

	vector<double> temp = { 0,0,0,0,0,0,0,0,0,0 };
	//vector<vector<double>> test = {{ 255/255.0,140/255.0,233/255.0,0/255.0,0/255.0,0/255.0,10/255.0,20/255.0,45/255.0,60/255.0}};
	//vector<vector<double>> test = {{ 255/255.0,140/255.0,233/255.0,0/255.0}};
	vector<vector<double>> train_encoders;
	for (int i = 0;i < train_lbls.size();i++) {
		temp[train_lbls[i][0]] = 1;
		train_encoders.push_back(temp);
		temp[train_lbls[i][0]] = 0;
	}
	double** d_train_imgs, **d_train_encoders, **d_test;
	//cudaAllocate2dOffVectorHostRef(&d_test, test);
	cudaAllocate2dOffVectorHostRef(&d_train_imgs, train_imgs);
	cudaAllocate2dOffVectorHostRef(&d_train_encoders, train_encoders);

	vector<vector<double>> x_batches = batchify(&train_imgs, batchSize);
	vector<vector<double>> y_batches = batchify(&train_encoders, batchSize);
	double** d_x_batches, **d_y_batches;
	cudaAllocate2dOffVectorHostRef(&d_x_batches, x_batches);
	cudaAllocate2dOffVectorHostRef(&d_y_batches, y_batches);


	int size = test_imgs[0].size();
	
	clock_t gpu_start, gpu_end;
	vector<double> out;
	vector<double> o;


	clock_t start, end;

	double l;

	l = 0.0;
	//loss = 0.0;
	hipMemcpy(&(*mlpara->d_weights_href[2]), &mlpara->h_weights[2][0][0], sizeof(double), hipMemcpyHostToDevice);

	gpu_start = clock();	
	for (int j = 0;j < 4;j++) {
		for (int i = 0;i < x_batches.size();i++) {
			//vector<vector<double>> cleanOut(batchSize, vector<double>(10));
			//for (int k = i * batchSize;k < (i + 1) * batchSize;k++) {
			//for (int k = i * batchSize;k < i * batchSize + 1;k++) {
				//printf("batch: %d\n", k);
				//cleanOut[k - i * batchSize] = mlpara->getCleanRun(d_train_imgs[k]);
			//}
			//vector<vector<double>> out = mlpar->getBatchRun(d_x_batches[i]);
			//compare2D(out, cleanOut);
			//for(int j = i * batchSize;j < batchSize;j++) l += mlpara->cleanerbp(d_train_imgs[j], d_train_encoders[j]);
			//for (int k = i * batchSize;k < (i + 1) * batchSize;k++) loss += mlpara->cleanerbp(d_train_imgs[k], d_train_encoders[k]);
			loss += mlpar->batchP(d_x_batches[i], d_y_batches[i]);
			//out = cudaCopyBatchBackToVectorHref(&mlpar->d_batch_errors_href[2], 10, batchSize);
			//compare3D(mlpar->h_weights, mlpara->h_weights);
			//loss += mlp->Wbp(train_imgs[i], train_encoders[i]);
			//cout << i << " : " << MSE << endl;
			if (i % (250 / batchSize) == 0) {
				gpu_end = clock();
				//cout << "Ground Example " << i << " error: " << l / (12 * batchSize) << endl;
				cout << "Epoch: " << j << ", Example " << i * batchSize << " error: " << loss / (250/ batchSize) << endl;
				cout << endl;
				l = 0.0;
				printExecution("Time taken", gpu_start, gpu_end);
				gpu_start = clock();
				loss = 0.0;
			}
		}
		printf("Epoch %d completed\n", j);
	}

	//loss = 0.0;
	//gpu_start = clock();
	//for (int j = 0;j < 4;j++) {
	//	for (int i = 0;i < train_imgs.size();i++) {
	//		loss += mlpara->cleanerbp(d_train_imgs[i], d_train_encoders[i]);
	//		if (i % 250 == 0) {
	//			gpu_end = clock();
	//			//cout << "Ground Example " << i << " error: " << l / (12 * batchSize) << endl;
	//			cout << "Epoch: " << j << ", Example " << i << " error: " << loss / (250) << endl;
	//			cout << endl;
	//			l = 0.0;
	//			printExecution("Time taken", gpu_start, gpu_end);
	//			gpu_start = clock();
	//			loss = 0.0;
	//		}
	//	}
	//}

	double** d_test_imgs = new double* [test_imgs.size()];
	cudaAllocate2dOffVectorHostRef(&d_test_imgs, test_imgs);

	double correct = 0.0;
    for (int i = 0;i < test_lbls.size();i++) {
		//vector<double> out = mlpara->getCleanRun(d_test_imgs[i]);
		vector<double> out = mlpar->getCleanRun(d_test_imgs[i]);
		//mlpar->batchRun(d_x_batches[i]);
		//vector<double> out = mlp->Wrun(test_imgs[i]);
		int ans = 0;
		double top = 0.0;
		for (int i = 0;i < 10;i++) 
			if (out[i] > top) {
			top = out[i];
			ans = i;
		}
		//cout << "image " << i << ": [";
		//for (int i = 0;i < 10;i++) cout << out[i] << ", ";
		//cout << "] " << ans << " : " << test_lbls[i][0] << endl;
		if (ans == test_lbls[i][0]) correct++;
    }
	double accuracy = correct / (double)test_lbls.size();
	printf("\n\nAccuracy ====== %f\n.... %f correct out of %d tests\n", accuracy, correct, test_lbls.size());



	return 0;
}

